#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <mma.h>
#include <hip/hip_fp16.h>

#include <vector>

namespace {
    __global__ void fast_jl_rademacher_cuda_kernel(
            float* __restrict__ output) {
        hiprandStateXORWOW_t random_state;
        hiprand_init(0, 0, 5, &random_state);
        output[0] = (float) hiprand(&random_state);
    }
} // namespace

void fast_jl_rademacher_cuda(
        torch::Tensor input,
        uint32_t seed,
        torch::Tensor output) {
    fast_jl_rademacher_cuda_kernel<<<2, 2>>>(output.data<float>());
}